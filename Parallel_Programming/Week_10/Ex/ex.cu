#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#define N 5

__global__ void CUDAStrCopy(char *A, char C[N])
{
	int i = threadIdx.x;
	C[i] = A[i] - 32;
	printf("%c\t", C[i]);
}

int main()
{
	char A[N];
	char C[N];
	char *pa, *pc;

	for(int i = 0; i < N; i++)
		A[i] = 'a';
	printf("C = \n");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMalloc((void**)&pa, N * sizeof(char));
	hipMalloc((void**)&pc, N * sizeof(char));
	hipMemcpy(pa, A, N * sizeof(char), hipMemcpyHostToDevice);
	hipError_t error = hipGetLastError();

	if(error != hipSuccess)
		printf("CUDA Error1: %s\n", hipGetErrorString(error));
	CUDAStrCopy<<<1, N>>>(pa, pc);
	error = hipGetLastError();

	if(error != hipSuccess)
		printf("CUDA Error2: %s\n", hipGetErrorString(error));

	hipMemcpy(C, pc, N * sizeof(char), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	int i;
	/*
		printf("Value of C in host after kernel execution\n");
		for(int i = 0; i < N; i++)
			printf("%c\n", C[i]);
	*/
	printf("Time taken = %f", elapsedTime);
	hipFree(pa);
	hipFree(pc);
	printf("\n");

	return 0;
}
