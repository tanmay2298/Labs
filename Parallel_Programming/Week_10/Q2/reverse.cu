#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 100

__global__ void CUDAStrCopy(char *str, int *len)
{
	int i = blockIdx.x;

	char temp = str[i];
	str[i] = str[*len - i - 1];
	str[*len - i - 1] = temp;
}

int main()
{
	char str[N];
	printf("Enter the string: ");
	scanf("%s", str);

	char *pstr;
	int len = strlen(str);
	int *plen;
	char str2[len];

	hipMalloc((void**)&pstr, len * sizeof(char));
	hipMalloc((void**)&plen, sizeof(int));
	hipMemcpy(pstr, str, len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(plen, &len, sizeof(int), hipMemcpyHostToDevice);

	CUDAStrCopy<<<len, 1>>>(pstr, plen);

	hipMemcpy(str, pstr, len * sizeof(char), hipMemcpyDeviceToHost);

	printf("Reversed String is: %s\n", str);

	hipFree(pstr);
	hipFree(plen);

	printf("\n");

		return 0;
}
