
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 3
#define M 2
__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;

//	if(tid < N)
	for(int i = 0; i < M; i++)
		c[tid * M + i] = a[tid * M + i] + b[tid * M + i];
}

int main()
{
//	int *a, *b, *c;
	int a[M * N], b[M * N], c[M * N];

	// host copies of variables a, b & c
	int *d_a, *d_b, *d_c;

	// device copies of variables a, b & c
	int size = sizeof(int) * M * N;

//	a = (int *)malloc(sizeof(int) * N);
//	b = (int *)malloc(sizeof(int) * N);
//	c = (int *)malloc(sizeof(int) * N);

	// Allocate space for device copies a, b & c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	// Setup input values
	printf("Enter values for a: ");
	for(int i = 0; i < M; i++)
		for(int j = 0; j < N; j++)
			scanf("%d", &a[i * N + j]);

	printf("Enter values for b: ");
	for(int i = 0; i < M; i++)
		for(int j = 0; j < N; j++)
			scanf("%d", &b[i * N + j]);

	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<1, M>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// print result
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
			printf("%d + %d = %d\n", a[i * N + j], b[i * N + j], c[i * N + j]);
	}

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
